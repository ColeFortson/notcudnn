#include "layer.h"

struct layer *layer_create(int ro, int co, int bs)
{
        struct layer *l;
        hipMallocManaged(&l, sizeof(struct layer));
        hipMalloc((void **)&(l->w), sizeof(float) * ro * co);
        hipMalloc((void **)&(l->b), sizeof(float) * ro * bs);
        hipMalloc((void **)&(l->wi), sizeof(float) * ro * bs);
        hipMalloc((void **)&(l->a), sizeof(float) * ro * bs);
        hipMalloc((void **)&(l->wu), sizeof(float) * ro * co);
        hipMalloc((void **)&(l->bu), sizeof(float) * ro * bs);
        l->rows = ro;
        l->cols = co;

        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
        hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
        hiprandGenerateUniform(gen, l->w, ro * co);
        hiprandGenerateUniform(gen, l->b, ro * bs);
        hiprandDestroyGenerator(gen);

        return l;
}

void layer_destroy(struct layer *l)
{
        hipFree(l->bu);
        hipFree(l->wu);
        hipFree(l->a);
        hipFree(l->wi);
        hipFree(l->b);
        hipFree(l->w);
        hipFree(l);
}
